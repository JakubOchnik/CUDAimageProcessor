#include "hip/hip_runtime.h"
#include "brightnessKernel.h"

void executeBrightnessKernel(Img* image, int shift, GPUcontroller* GPU) {
	dim3 grid(image->getResolutionW(), image->getResolutionH());
	int channels = image->getChannelNum();
	int width = image->getResolutionW();
	int height = image->getResolutionH();
	size_t size = channels * width * height * sizeof(unsigned char);
	calculateBrightness << <grid, 1 >> > (GPU->getImgPtr(), channels, shift);

	hipMemcpy(image->getImg()->data, GPU->getImgPtr(), size, hipMemcpyDeviceToHost);
}

__global__ void calculateBrightness(unsigned char* image, int channels, int shift) {
	int x = blockIdx.x;
	int y = blockIdx.y;

	int index = (x + y * gridDim.x) * channels;

	for (int i = 0; i < channels; i++) {
		int outputPixel = image[index + i] + shift;
		if (outputPixel > 255)
			image[index + i] = 255;
		else if (outputPixel < 0)
			image[index + i] = 0;
		else
			image[index + i] = (char)outputPixel;
	}
}