#include "hip/hip_runtime.h"
#include "brightnessKernel.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void calculateBrightness(unsigned char* image, int channels, int width, int height, int shift);

void executeBrightnessKernel(Img* image, int shift, GPUcontroller* GPU) {
	dim3 grid(image->getResolutionW(), image->getResolutionH());
	int channels = image->getChannelNum();
	int width = image->getResolutionW();
	int height = image->getResolutionH();
	size_t size = channels * width * height * sizeof(unsigned char);
	calculateBrightness<<<grid, 1>>>(GPU->getImgPtr(), channels, width, height, shift);
	hipMemcpy(image->getImg()->data, GPU->getImgPtr(),size, hipMemcpyDeviceToHost);
	//hipDeviceSynchronize();
	//printf("");
}

__global__ void calculateBrightness(unsigned char* image, int channels, int width, int height, int shift) {
	int x = blockIdx.x;
	int y = blockIdx.y;

	int index = (x + y * gridDim.x) * channels;
	
	for (int i = 0; i < channels; i++) {
		//image[index + i] += shift;
		int outputPixel = image[index + i] + shift;
		// debug:
		/*if (index < 100)
			printf("%d input: %d outputPixel: %d\n ", index, image[index], outputPixel);*/
		if (outputPixel > 255)
			image[index + i] = 255;
		else if (outputPixel < 0)
			image[index + i] = 0;
		else
			image[index + i] = (char)outputPixel;
	}
	// debug:
	/*if(index<100)
		printf("index %d out %d shift %d\n", index, image[index], shift);
	image[index] = 1;*/
}