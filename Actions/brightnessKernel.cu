#include "hip/hip_runtime.h"
#include "brightnessKernel.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void calculateBrightness(unsigned char* image, int channels, int width, int height, int shift);

void executeBrightnessKernel(Img* image, int shift) {
	dim3 grid(image->getResolutionW(), image->getResolutionH());
	int channels = image->getChannelNum();
	int width = image->getResolutionW();
	int height = image->getResolutionH();
	calculateBrightness<<<grid, 1>>>(image->getImg()->data, channels, width, height, shift);
	hipDeviceSynchronize();
	printf("");
}

__global__ void calculateBrightness(unsigned char* image, int channels, int width, int height, int shift) {
	int x = blockIdx.x;
	int y = blockIdx.y;

	int index = (x + y * gridDim.x) * channels;

	for (int i = 0; i < channels; i++) {
		image[index + i] += shift;
		int outputPixel = (int)image[index + i] + shift;
		// clipping:

		if (outputPixel > 255)
			image[index + i] = 255;
		else if (outputPixel < 0)
			image[index + i] = 0;
		else
			image[index + i] = (char)outputPixel;
	}
}