#include "hip/hip_runtime.h"
﻿#include "contrastKernel.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void calculateContrast(unsigned char* image, int channels, float factor);
__device__ unsigned char truncate(float value);
void executeContrastKernel(Img* image, int value, GPUcontroller* GPU) {
	dim3 grid(image->getResolutionW(), image->getResolutionH());
	int channels = image->getChannelNum();
	int width = image->getResolutionW();
	int height = image->getResolutionH();
	size_t size = channels * width * height * sizeof(unsigned char);
	float factor = (259 * (value + 255)) / (float)(255 * (259 - value));
	calculateContrast <<<grid, 1 >>> (GPU->getImgPtr(), channels, factor);
	hipMemcpy(image->getImg()->data, GPU->getImgPtr(), size, hipMemcpyDeviceToHost);
}

__global__ void calculateContrast(unsigned char* image, int channels, float factor) {
	int x = blockIdx.x;
	int y = blockIdx.y;

	int index = (x + y * gridDim.x) * channels;

	for (int i = 0; i < channels; i++) {
		image[index+i] = truncate(factor*(image[index + i]-128)+128);
	}
}

__device__ unsigned char truncate(float value) {
	if (value > 255)
		return 255;
	else if (value < 0)
		return 0;
	return (unsigned char)value;
}