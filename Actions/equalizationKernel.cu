#include "hip/hip_runtime.h"
﻿#include "equalizationKernel.h"


bool executeEqualizationKernel(Img* image, GPUcontroller* GPU) {
	int channels = image->getChannelNum();
	int width = image->getResolutionW();
	int height = image->getResolutionH();
	int* min;
	int* max;
	// memory allocation with error checking
	min = new (std::nothrow) int[channels];
	if (!min) {
		return false;
	}
	max = new (std::nothrow) int[channels];
	if (!max) {
		return false;
	}
	for (int i = 0; i < channels; i++) {
		min[i] = 255;
		max[i] = 0;
	}
	int* dev_min = nullptr;
	int* dev_max = nullptr;
	hipMalloc((void**)&dev_min, channels * sizeof(int));
	hipMalloc((void**)&dev_max, channels * sizeof(int));

	hipMemcpy(dev_min, min, channels * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_max, max, channels * sizeof(int), hipMemcpyHostToDevice);

	dim3 grid(width, height);

	size_t size = channels * width * height * sizeof(unsigned char);
	calculateEdgeBrightness << <grid, 1 >> > (GPU->getImgPtr(), channels, dev_min, dev_max);
	hipMemcpy(min, dev_min, channels * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(max, dev_max, channels * sizeof(int), hipMemcpyDeviceToHost);
	calculateEqualization << <grid, 1 >> > (GPU->getImgPtr(), channels, dev_min, dev_max);

	hipMemcpy(image->getImg()->data, GPU->getImgPtr(), size, hipMemcpyDeviceToHost);
	hipFree(dev_min);
	hipFree(dev_max);
	delete[] min;
	delete[] max;
	return true;
}

__global__ void calculateEdgeBrightness(unsigned char* image, int channels, int* min, int* max) {
	int x = blockIdx.x;
	int y = blockIdx.y;
	int imageIdx = (x + y * gridDim.x) * channels;

	for (int i = 0; i < channels; i++) {
		atomicMin(&min[i], image[imageIdx + i]);
		atomicMax(&max[i], image[imageIdx + i]);
	}
}

__global__ void calculateEqualization(unsigned char* image, int channels, int* min, int* max) {
	int x = blockIdx.x;
	int y = blockIdx.y;

	int image_idx = (x + y * gridDim.x) * channels;

	for (int i = 0; i < channels; i++) {
		image[image_idx + i] = getEqualizedValue(image[image_idx + i], min[i], max[i]);
	}
}

__device__ int getEqualizedValue(unsigned char value, int min, int max) {
	int target_min = 0;
	int target_max = 255;

	return(target_min + (value - min) * (int)((target_max - target_min) / (max - min)));
}