#include "hip/hip_runtime.h"
﻿#include "invertionKernel.h"


void executeInvertionKernel(Img* image, GPUcontroller* GPU) {
	dim3 grid(image->getResolutionW(), image->getResolutionH());
	int channels = image->getChannelNum();
	int width = image->getResolutionW();
	int height = image->getResolutionH();
	size_t size = channels * width * height * sizeof(unsigned char);
	invertImage << <grid, 1 >> > (GPU->getImgPtr(), channels);
	hipMemcpy(image->getImg()->data, GPU->getImgPtr(), size, hipMemcpyDeviceToHost);
}

__global__ void invertImage(unsigned char* image, int channels) {
	int x = blockIdx.x;
	int y = blockIdx.y;

	int index = (x + y * gridDim.x) * channels;

	for (int i = 0; i < channels; i++)
		image[index + i] = 255 - image[index + i];
}