#include "hip/hip_runtime.h"
﻿#include "contrastKernel.h"

void executeContrastKernel(Img* image, const int value, GPUcontroller* GPU)
{
	dim3 grid(image->getResolutionW(), image->getResolutionH());
	const int channels = image->getChannelNum();
	const int width = image->getResolutionW();
	const int height = image->getResolutionH();
	const size_t size = channels * width * height * sizeof(unsigned char);
	const float factor = 259 * (value + 255) / static_cast<float>(255 * (259 - value));
	calculateContrast << <grid, 1 >> > (GPU->getImgPtr(), channels, factor);
	hipMemcpy(image->getImg()->data, GPU->getImgPtr(), size, hipMemcpyDeviceToHost);
}

__global__ void calculateContrast(unsigned char* image, int channels, float factor)
{
	int x = blockIdx.x;
	int y = blockIdx.y;

	int index = (x + y * gridDim.x) * channels;

	for (int i = 0; i < channels; i++)
	{
		image[index + i] = truncate(factor * (image[index + i] - 128) + 128);
	}
}

__device__ unsigned char truncate(float value)
{
	if (value > 255)
		return 255;
	if (value < 0)
		return 0;
	return static_cast<unsigned char>(value);
}
