#include "hip/hip_runtime.h"
#include <Commands/GpuImg/Kernels/BrightnessKernel.cuh>

void executeBrightnessKernel(Img &image, const int shift, GPUcontroller *GPU)
{
	dim3 grid(image.getResolutionW(), image.getResolutionH());
	const int channels = image.getChannelNum();
	const int width = image.getResolutionW();
	const int height = image.getResolutionH();
	const size_t size = channels * width * height * sizeof(unsigned char);
	calculateBrightness<<<grid, 1>>>(image.getImg()->data, channels, shift);

	//hipMemcpy(image.getImg()->data, GPU->getImgPtr(), size, hipMemcpyDeviceToHost);
}

__global__ void calculateBrightness(unsigned char *image, int channels, int shift)
{
	int x = blockIdx.x;
	int y = blockIdx.y;

	int index = (x + y * gridDim.x) * channels;

	for (int i = 0; i < channels; i++)
	{
		int outputPixel = image[index + i] + shift;
		if (outputPixel > 255)
			image[index + i] = 255;
		else if (outputPixel < 0)
			image[index + i] = 0;
		else
			image[index + i] = static_cast<char>(outputPixel);
	}
}
