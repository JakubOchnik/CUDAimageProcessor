#include "hip/hip_runtime.h"
﻿#include <Commands/GpuImg/Kernels/EqualizationKernel.cuh>

bool executeEqualizationKernel(Img& image, GPUcontroller* GPU)
{
	const unsigned int channels = image.getChannelNum();
	const unsigned int width = image.getResolutionW();
	const unsigned int height = image.getResolutionH();
	// memory allocation with error checking
	auto min = new(std::nothrow) int[channels];
	if (!min)
	{
		return false;
	}
	auto max = new(std::nothrow) int[channels];
	if (!max)
	{
		return false;
	}
	for (int i = 0; i < channels; i++)
	{
		min[i] = 255;
		max[i] = 0;
	}
	int* devMin = nullptr;
	int* devMax = nullptr;
	hipMalloc((void**)&devMin, channels * sizeof(int));
	hipMalloc((void**)&devMax, channels * sizeof(int));

	hipMemcpy(devMin, min, channels * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(devMax, max, channels * sizeof(int), hipMemcpyHostToDevice);

	dim3 grid(width, height);

	const size_t size = channels * width * height * sizeof(unsigned char);
	calculateEdgeBrightness << <grid, 1 >> > (GPU->getImgPtr(), channels, devMin, devMax);
	hipMemcpy(min, devMin, channels * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(max, devMax, channels * sizeof(int), hipMemcpyDeviceToHost);
	calculateEqualization << <grid, 1 >> > (GPU->getImgPtr(), channels, devMin, devMax);

	hipMemcpy(image.getImg()->data, GPU->getImgPtr(), size, hipMemcpyDeviceToHost);
	hipFree(devMin);
	hipFree(devMax);
	delete[] min;
	delete[] max;
	return true;
}

__global__ void calculateEdgeBrightness(unsigned char* image, int channels, int* min, int* max)
{
	int x = blockIdx.x;
	int y = blockIdx.y;
	int imageIdx = (x + y * gridDim.x) * channels;

	for (int i = 0; i < channels; i++)
	{
		atomicMin(&min[i], image[imageIdx + i]);
		atomicMax(&max[i], image[imageIdx + i]);
	}
}

__global__ void calculateEqualization(unsigned char* image, int channels, int* min, int* max)
{
	int x = blockIdx.x;
	int y = blockIdx.y;

	int image_idx = (x + y * gridDim.x) * channels;

	for (int i = 0; i < channels; i++)
	{
		image[image_idx + i] = getEqualizedValue(image[image_idx + i], min[i], max[i]);
	}
}

__device__ int getEqualizedValue(unsigned char value, int min, int max)
{
	int target_min = 0;
	int target_max = 255;

	return (target_min + (value - min) * ((target_max - target_min) / (max - min)));
}
