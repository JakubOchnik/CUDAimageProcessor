#include "hip/hip_runtime.h"
﻿#include <Commands/GpuImg/Kernels/InvertionKernel.cuh>


void executeInvertionKernel(Img& image, const std::shared_ptr<GPUcontroller>& GPU)
{
	dim3 grid(image.getResolutionW(), image.getResolutionH());
	const int channels = image.getChannelNum();
	const int width = image.getResolutionW();
	const int height = image.getResolutionH();
	const size_t size = channels * width * height * sizeof(unsigned char);
	invertImage << <grid, 1 >> > (GPU->getImgPtr(), channels);

	hipMemcpy(image.getImg().data, GPU->getImgPtr(), size, hipMemcpyDeviceToHost);
}

__global__ void invertImage(unsigned char* image, int channels)
{
	int x = blockIdx.x;
	int y = blockIdx.y;

	int index = (x + y * gridDim.x) * channels;

	for (int i = 0; i < channels; i++)
		image[index + i] = 255 - image[index + i];
}
